
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__global__ void compute_acc1(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	float diffx, diffy, diffz;

	for(int j = 0; j < n_particules; j++)
	{
		if(i != j)
		{
			diffx = positionsGPU[j].x - positionsGPU[i].x;
			diffy = positionsGPU[j].y - positionsGPU[i].y;
			diffz = positionsGPU[j].z - positionsGPU[i].z;

			float dij = diffx * diffx + diffy * diffy + diffz * diffz;

			if (dij < 1.0)
				{
					dij = 10.0;
				}
				else
				{
					dij = std::sqrt(dij);
					dij = 10.0 / (dij * dij * dij);
				}

			accelerationsGPU[i].x += diffx * dij * massesGPU[j];
			accelerationsGPU[i].y += diffy * dij * massesGPU[j];
			accelerationsGPU[i].z += diffz * dij * massesGPU[j];
		}
	}

}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	velocitiesGPU[i].x += accelerationsGPU[i].x * 2.0f;
	velocitiesGPU[i].y += accelerationsGPU[i].x * 2.0f;
	velocitiesGPU[i].z += accelerationsGPU[i].x * 2.0f;
	positionsGPU[i].x += velocitiesGPU[i].x * 0.1f;
	positionsGPU[i].y += velocitiesGPU[i].y * 0.1f;
	positionsGPU[i].z += velocitiesGPU[i].z * 0.1f;
}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU